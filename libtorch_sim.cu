#include <memory>
#include <vector>
#include <iostream>
#include <fstream>
#include <cstring>
#include <cassert>
#include <cmath>
#include <sys/time.h>
#include <stdio.h>
#include <omp.h>
#include "wtime.h"
#include "herror.h"
#include "sim.cuh"
#include<torch/script.h>
using namespace std;
#define NO_MEAN
#define GPU
#define WARP
//#define DEBUG
//#define Stride_width 10
//#define Total_Trace 1024

Tick Num = 0;


float *read_numbers(char *fname, int sz)
{
  float *ret = new float[sz];
  ifstream in(fname);
  //printf("Trying to read from %s\n", fname);
  for (int i = 0; i < sz; i++)
    in >> ret[i];
  return ret;
}



int main(int argc, char *argv[])
{
  //printf("args count: %d\n", argc);
#ifdef WARMUP
  if (argc != 8)
  {
    cerr << "Usage: ./simulator_q <trace> <aux trace> <lat module> <Total trace><# inst> <W (warmup)> <U>" << endl;
    return 0;
    //int W= atoi(argv[6]);
  }
#else
  if (argc != 8)
  {
    cerr << "Usage: ./simulator_qq <trace> <aux trace> <lat module> <Total trace> <#Insts> <W not used> <U>" << endl;
  return 0;
}
#endif
//int arg_idx = 4;
//float *varPtr = read_numbers(argv[arg_idx++], TD_SIZE);

//cout<< argv[3] << endl;
torch::jit::script::Module lat_module;
  try {
    // Deserialize the ScriptModule from a file using torch::jit::load().
    lat_module = torch::jit::load(argv[3]);
#ifdef GPU
    lat_module.to(torch::kCUDA);
#endif
  }
  catch (const c10::Error& e) {
    cerr << "error loading the model\n";
    return 0;
  }
//lat_module.save("libtorch.pt");
//return 0;
//cout<<endl;
const unsigned long long int Total_Trace = atoi(argv[4]);
const unsigned long long int Instructions = atoi(argv[5]);
//cout<< "Total_Trace: "<< Total_Trace << ", Instructions: "<< Instructions << endl;
//std::string model_path(argv[3]);
int N= 10;
at::Tensor input = torch::ones({atoi(argv[4]), ML_SIZE});
float *inp= input.data_ptr<float>();
//cout<<"Input dims: "<< input_dims << ", output dims: "<<output_dims << endl;
float *inputPtr,*inputPtr1, *inputPtr2, *output;
H_ERR(hipMalloc((void **)&inputPtr, sizeof(float) * (ML_SIZE + TD_SIZE*(Stream_width-1)) * Total_Trace));
hipMemset(inputPtr, 0, ML_SIZE + N*Stream_width);
H_ERR(hipMalloc((void **)&inputPtr1, sizeof(float) * (ML_SIZE + TD_SIZE*(Stream_width-1)) * Total_Trace));
H_ERR(hipMalloc((void **)&inputPtr2, sizeof(float) * (ML_SIZE + TD_SIZE*(Stream_width-1)) * Total_Trace));
H_ERR(hipMalloc((void **)&output, sizeof(float) * Total_Trace * 33));
//cout<< "Input dim: "<< ML_SIZE * Total_Trace << endl;
float *stream;
stream = (float *)malloc(Stream_width * TD_SIZE * sizeof(float));
float *trace;
Tick *aux_trace;
//trace = (float *)malloc(TRACE_DIM * Instructions * sizeof(float));
//aux_trace = (Tick *)malloc(AUX_TRACE_DIM * Instructions * sizeof(Tick));
int Batch_size = Instructions / Total_Trace;
//printf("Batchsize: %d\n",Batch_size);
if(Instructions%Total_Trace!=0){
        printf("Prev bsize: %d, mew bsize: %d\n", Batch_size, Batch_size + 1);
        Batch_size= Batch_size +1;
        unsigned long long int new_instr=  (Batch_size+1)*Total_Trace;
        trace = (float *)malloc(TRACE_DIM * new_instr * sizeof(float));
        aux_trace = (Tick *)malloc(AUX_TRACE_DIM * new_instr* sizeof(Tick));
        unsigned long long int index= Instructions;
        for (; index<new_instr; index++){
                memcpy(&trace[index * TRACE_DIM], zeros, sizeof(float)*TRACE_DIM);
                memcpy(&aux_trace[index * AUX_TRACE_DIM], zeros, sizeof(Tick)*AUX_TRACE_DIM);
                index+=1;
         }
}
else{
        trace = (float *)malloc(TRACE_DIM * Instructions * sizeof(float));
        aux_trace = (Tick *)malloc(AUX_TRACE_DIM * Instructions * sizeof(Tick));}
read_trace_mem(argv[1], argv[2], trace, aux_trace, Instructions);
//omp_set_num_threads(1);
double measured_time = 0.0;
int *fetched_inst_num = new int[Total_Trace];
int *fetched = new int[Total_Trace];
int *ROB_flag = new int[Total_Trace];
float *trace_all[Total_Trace];
Tick *aux_trace_all[Total_Trace];
int index_all[Total_Trace];
int *index_all_gpu;
H_ERR(hipMalloc((void **)&index_all_gpu, sizeof(int) * Total_Trace));
//printf("variable init\n");
int W=0;
#ifdef WARMUP
W= atoi(argv[6]);
#endif
int U= atoi(argv[7]);
//#pragma omp parallel for
for (int i = 0; i < Total_Trace; i++)
{
  long long int offset = (i * (Batch_size))-U;
#ifdef WARMUP
    offset= offset - W;
    //cout<< "W: "<<W<<", Index: "<< i <<", Offset: "<< offset << endl; 
#endif
    if(offset<0){offset=0;}
    //cout<< "W: "<<W<<", Index: "<< i <<" ,start: "<< offset << " ,warmup: "<< offset-W<< " ,End: "<<(offset + Batch_size) << endl;
  //if(offset>Instructions)printf("Index: %d, offset: %d\n",i,offset);
  //assert(offset<=Instructions);
  //assert(offset>=0);
  index_all[i]= offset;
  //cout<< "W: "<<W<<", Index: "<< i <<",Start: "<< offset <<" ,warmup: "<<offset + W << "End: "<<offset + Batch_size + W << endl;
  trace_all[i]= trace + offset * TRACE_DIM;
  aux_trace_all[i]= aux_trace + offset * AUX_TRACE_DIM;
}
// printf("Allocated. \n");
//return 0;
float *default_val_d;
Tick *curTick, *lastFetchTick;
int *status;
H_ERR(hipMalloc((void **)&curTick, sizeof(Tick) * Total_Trace));
H_ERR(hipMalloc((void **)&lastFetchTick, sizeof(Tick) * Total_Trace));
H_ERR(hipMalloc((void **)&status, sizeof(int) * Total_Trace));
hipMemset(curTick, 0, Total_Trace);
hipMemset(lastFetchTick, 0, Total_Trace);
hipMemset(status, 1, Total_Trace);
struct SQ *sq= new SQ[Total_Trace];
struct ROB *rob= new ROB[Total_Trace];
struct Inst *inst= new Inst[Total_Trace];
struct ROB *rob_d; 
struct SQ *sq_d;
struct Inst *inst_d;
H_ERR(hipMalloc((void **)&rob_d, sizeof(ROB)*Total_Trace));
H_ERR(hipMalloc((void **)&sq_d, sizeof(SQ)*Total_Trace));
H_ERR(hipMalloc((void **)&inst_d, sizeof(Inst)*Total_Trace));
// For factor, mean and default values
H_ERR(hipMalloc((void **)&default_val_d, sizeof(float) * (TD_SIZE)));
H_ERR(hipMemcpy(default_val_d, &default_val, sizeof(float) * TD_SIZE, hipMemcpyHostToDevice));
struct timeval check3, t, total_start, total_end;
int iteration = 0;
gettimeofday(&total_start, NULL);
double start_ = wtime();
double red=0, tr=0,upd=0;
FILE *pFile;
pFile= fopen ("libcustom.bin", "wb");
//outFile= fopen("pred.bin", "wb");
//printf("Simulation started.. \n");
//:return 0;
int total_iterations= Batch_size + W;
initialization<<<4096,32>>>(rob_d,Total_Trace);
H_ERR(hipDeviceSynchronize());
int N_flag=0;
inputPtr= inputPtr1;
float *next= inputPtr2;
double count=0;
//**********************First copy
while (iteration < total_iterations){
  {cout << "\nIteration: " << iteration << endl;}
    N_flag= iteration%Stream_width;
    //printf("N flag: %d, current: %p, next: %p\n",N_flag, inputPtr, next);
if(N_flag==0){
  double st= wtime();
  if (iteration!=0){
  	printf("**shifted**\n");
	shift<<<4096, 64>>>(inputPtr, next, rob_d, Total_Trace);
	float *temp_inp= inputPtr;
  	inputPtr= next;
  	next= temp_inp;
  }
  printf("Copied\n");
      #pragma omp parallel for
  for (int i = 0; i < Total_Trace; i++)
  {
    index_all[i]+= Stream_width;
    //if (!inst[i].read_sim_mem(trace_all[i], aux_trace_all[i],index_all[i]))
    if (!inst[i].batched_copy(trace_all[i], aux_trace_all[i], stream , index_all[i]))
    {cout << "Error\n";}
    //display(stream, TD_SIZE, Stream_width);
    trace_all[i] += (Stream_width * TRACE_DIM); 
    aux_trace_all[i] += (Stream_width * AUX_TRACE_DIM);
  }
  double check1 = wtime();
  red+= (check1-st);
  H_ERR(hipMemcpy(inst_d, inst, sizeof(Inst) * Total_Trace, hipMemcpyHostToDevice));
  H_ERR(hipMemcpy(inputPtr, stream, sizeof(float) * TD_SIZE * Stream_width, hipMemcpyHostToDevice));
  //printf("inpt: %p, stream: %p\n", inputPtr, stream);
  double check2 = wtime();
  count+= (check2-check1);
  //cout<<"Data transferred\n";
  H_ERR(hipMemcpy(index_all_gpu, index_all, sizeof(int) * Total_Trace, hipMemcpyHostToDevice));
  gettimeofday(&t, NULL);
}
  printf("N flag: %d, current: %p, next: %p\n",N_flag, inputPtr, next);
  // **********************************Start other for loop from here *************************************************
  preprocess<<<4096, 32>>>(rob_d,sq_d,inst_d, default_val_d, inputPtr, status, Total_Trace, index_all_gpu, iteration, W, Batch_size,N_flag);
  H_ERR(hipDeviceSynchronize());
  //cout<<"Preprocess done \n"<<endl; 
  //double check3= wtime();
  gettimeofday(&check3, NULL);
  N_flag= iteration%N;
  printf("N flag: %d\n",N_flag);
  H_ERR(hipMemcpy(inp,inputPtr, sizeof(float) * ML_SIZE * Total_Trace, hipMemcpyDeviceToHost));
  //fwrite(inp, sizeof(float), ML_SIZE*Total_Trace, pFile);
  //int *in= (int *) malloc(1);
  //in[0]=iteration; 
  //fwrite(in,sizeof(int),1,pFile);
  //printf("Input:\n");
  //display(inp, 51,4);
  //pre+= (check3-t);
  //printf(",%f \n",(check3-t));
  //check3 = wtime();
  //pre+= (check3-check2);
  
  std::vector<torch::jit::IValue> inputs;
  inputs.push_back(input.cuda());  
  at::Tensor outputs = lat_module.forward(inputs).toTensor();
  outputs=outputs.to(at::kCPU);
  hipStreamSynchronize(0);
  //cout<<outputs<<endl;
  double check4= wtime();
  //inf+= (check4-check3);
  //cout<<"Output size: "<< outputs.sizes()[0]<<endl;
  //cout<<"Inference done \n";
  //int out_shape= outputs.sizes()[1];
  int out_shape=33;
  //float *output;
  //H_ERR(hipMalloc((void **)&output, sizeof(float)*33));
  //H_ERR(hipMemcpy(output, outputs.data_ptr<float>(), sizeof(float)*Total_Trace*out_shape, hipMemcpyHostToDevice));
  update<<<4096,32>>>(rob_d,sq_d, inputPtr, output, status, Total_Trace, out_shape, iteration, W, Batch_size, index_all_gpu);
  H_ERR(hipDeviceSynchronize());
  //cout<<"Update done\n";
  double check5=wtime();
  upd+=(check5-check4);
  iteration++;
  //if(iteration==11) return 0;
}
double e = wtime();
double end_ = wtime();
double total_time = check3.tv_sec - t.tv_sec + (check3.tv_usec - t.tv_usec) / 1000000.0;
printf("Avg: Total: %f, Pre %f\n",(end_-start_)/Instructions,total_time/Instructions);
printf("Time: %f\n", count/iteration);
return 0;
fclose(pFile);
//fclose(outFile);
//printf("%.4f, %.4f, %.4f, %.4f, %.4f\n",red, tr, pre, inf, upd);
//double end_ = wtime();

gettimeofday(&total_end, NULL);
Tick *total_tick;
H_ERR(hipMalloc((void **)&total_tick, sizeof(Tick)));
string p(argv[3]);
string d(argv[2]);
size_t found= p.find_last_of("/\\");
//size_t found1=p.find_last_of("_");
cout<<argv[0]<<",";
cout<<p.substr(found+1)<<",";
found= d.find_last_of("/\\");
cout<< d.substr(found+1) <<",";
printf("%llu,%llu,%d,%d,",Instructions,Total_Trace,W,iteration);
result<<<1, 1>>>(rob_d, Total_Trace, Instructions, total_tick);
H_ERR(hipDeviceSynchronize());
//H_ERR(hipMemcpy(&total_tick[i], total_tick_d[i], sizeof(Tick), hipMemcpyDeviceToHost));
total_time = total_end.tv_sec - total_start.tv_sec + (total_end.tv_usec - total_start.tv_usec) / 1000000.0;
cout <<total_time<< endl;
printf("Time: %f\n", count);
return 0;
cout << Instructions << " instructions finish by " << (curTick - 1) << "\n";
cout << "Time: " << total_time << "\n";
cout << "MIPS: " << Instructions / total_time / 1000000.0 << "\n";
cout << "USPI: " << total_time * 1000000.0 / Instructions << "\n";
cout << "Measured Time: " << measured_time / Instructions << "\n";
//cout << "Cases: " << Case0 << " " << Case1 << " " << Case2 << " " << Case3 << " " << Case4 << " " << Case5 << "\n";
cout << "Trace: " << argv[1] << "\n";
#ifdef CLASSIFY
cout << "Model: " << argv[3] << " " << argv[4] << "\n";
#else
  //cout << "Lat Model: " << argv[3] << "\n";
#endif
return 0;
}
